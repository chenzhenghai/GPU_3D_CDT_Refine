#include "CudaThrust.h"
#include "thrust/execution_policy.h"

int updateActiveListByMarker_Slot
(
	IntD	    &t_marker,
	IntD		&t_active,
	int         numberofelements
)
{
	thrust::counting_iterator<int> first(0);
	thrust::counting_iterator<int> last = first + numberofelements;

	t_active.resize(numberofelements);

	t_active.erase(
		thrust::copy_if(
			first,
			last,
			t_marker.begin(),
			t_active.begin(),
			isNotNegativeInt()),
		t_active.end());

	return t_active.size();
}

int updateActiveListByMarker_Slot
(
	ByteD	    &t_marker,
	IntD		&t_active,
	int         numberofelements
)
{
	thrust::counting_iterator<int> first(0);
	thrust::counting_iterator<int> last = first + numberofelements;

	t_active.resize(numberofelements);

	t_active.erase(
		thrust::copy_if(
			first,
			last,
			t_marker.begin(),
			t_active.begin(),
			isNotNegativeByte()),
		t_active.end());

	return t_active.size();
}

int updateActiveListByStatus_Val
(
	IntD		&t_input,
	TriStatusD	&t_status,
	IntD		&t_active,
	int			numberofelements
)
{
	thrust::counting_iterator<int> first(0);
	thrust::counting_iterator<int> last = first + numberofelements;

	t_active.resize(numberofelements);

	t_active.erase(
		thrust::copy_if(
			t_input.begin(),
			t_input.end(),
			t_status.begin(),
			t_active.begin(),
			isNotEmptyTri()),
		t_active.end());

	return t_active.size();
}

int updateActiveListByStatus_Slot
(
	TetStatusD	&t_status,
	IntD		&t_active,
	int			numberofelements
)
{
	thrust::counting_iterator<int> first(0);
	thrust::counting_iterator<int> last = first + numberofelements;

	t_active.resize(numberofelements);

	t_active.erase(
		thrust::copy_if(
			first,
			last,
			t_status.begin(),
			t_active.begin(),
			isBadTet()),
		t_active.end());

	return t_active.size();
}

int updateEmptyTets
(
	TetStatusD	&t_tetstatus,
	IntD	&t_emptytets
)
{
	const int tetlistsize = t_tetstatus.size();

	t_emptytets.resize(tetlistsize);

	thrust::counting_iterator<int> first(0);
	thrust::counting_iterator<int> last(tetlistsize);

	t_emptytets.erase(
		thrust::copy_if(
			first,
			last,
			t_tetstatus.begin(),
			t_emptytets.begin(),
			isEmptyTet()),
		t_emptytets.end());

	return t_emptytets.size();
}

int updateEmptyTris
(
	TriStatusD &t_tristatus,
	IntD	&t_emptytris
)
{
	const int trilistsize = t_tristatus.size();

	t_emptytris.resize(trilistsize);

	thrust::counting_iterator<int> first(0);
	thrust::counting_iterator<int> last(trilistsize);

	t_emptytris.erase(
		thrust::copy_if(
			first,
			last,
			t_tristatus.begin(),
			t_emptytris.begin(),
			isEmptyTri()),
		t_emptytris.end());

	return t_emptytris.size();
}

void gpuMemoryCheck()
{
	size_t free_byte;
	size_t total_byte;
	hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);
	if (hipSuccess != cuda_status)
	{
		printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status));
		exit(1);
	}
	double free_db = (double)free_byte;
	double total_db = (double)total_byte;
	double used_db = total_db - free_db;
	printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",
		used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}
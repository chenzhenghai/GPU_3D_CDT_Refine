#include "hip/hip_runtime.h"
#include "CudaSplitBadElement.h"
#include "CudaInsertPoint.h"
#include "CudaMesh.h"
#include <time.h>

// This function splits the encroached segments iteratively
void splitBadElements(
	RealD& t_pointlist,
	TriHandleD& t_point2trilist,
	TetHandleD& t_point2tetlist,
	PointTypeD& t_pointtypelist,
	RealD& t_pointradius,
	IntD& t_seglist,
	TriHandleD& t_seg2trilist,
	TetHandleD& t_seg2tetlist,
	IntD& t_seg2parentidxlist,
	IntD& t_segparentendpointidxlist,
	TriStatusD& t_segstatus,
	IntD& t_trifacelist,
	TetHandleD& t_tri2tetlist,
	TriHandleD& t_tri2trilist,
	TriHandleD& t_tri2seglist,
	IntD& t_tri2parentidxlist,
	IntD& t_triid2parentoffsetlist,
	IntD& t_triparentendpointidxlist,
	TriStatusD& t_tristatus,
	IntD& t_tetlist,
	TetHandleD& t_neighborlist,
	TriHandleD& t_tet2trilist,
	TriHandleD& t_tet2seglist,
	TetStatusD& t_tetstatus,
	IntD& t_segencmarker,
	IntD& t_subfaceencmarker,
	int& numofpoints,
	int& numofsubseg,
	int& numofsubface,
	int& numoftet,
	MESHBH* behavior,
	int debug_msg,
	bool debug_error,
	bool debug_timing
)
{
	IntD t_badeleidlist;
	IntD t_encseglist, t_encsubfacelist, t_badtetlist;
	IntD t_threadmarker;

	int numberofbadeles;
	int numberofencsegs, numberofencsubfaces, numberofbadtets;

	clock_t tv[2];
	int npt[2];
	int code = 1;
	int iteration = 0;
	int counter;
	while (true)
	{
		//printf("%d, ", iteration);
		// Update the active bad elements list.
		// Exclude the empty ones (their encroachment markers have already been set to -1).
		numberofencsegs = updateActiveListByMarker_Slot(t_segencmarker, t_encseglist, numofsubseg);
		numberofencsubfaces = updateActiveListByMarker_Slot(t_subfaceencmarker, t_encsubfacelist, numofsubface);
		numberofbadtets = updateActiveListByStatus_Slot(t_tetstatus, t_badtetlist, numoftet);
		if (numberofbadtets == 0)
			break;
		if (numberofbadtets <= behavior->minbadtets && iteration >= behavior->miniter)
		{
			code = 0;
			break;
		}

		numberofbadeles = numberofencsegs + numberofencsubfaces + numberofbadtets;
		if (debug_msg) printf("      Iteration #%d: number of bad elements = %d (#%d segs, #%d subfaces, #%d tets)\n", 
			iteration, numberofbadeles, numberofencsegs, numberofencsubfaces, numberofbadtets);

		t_badeleidlist.resize(numberofbadeles);
		thrust::copy_n(t_encseglist.begin(), numberofencsegs, t_badeleidlist.begin());
		thrust::copy_n(t_encsubfacelist.begin(), numberofencsubfaces, t_badeleidlist.begin() + numberofencsegs);
		thrust::copy_n(t_badtetlist.begin(), numberofbadtets, t_badeleidlist.begin() + numberofencsegs + numberofencsubfaces);

		t_threadmarker.resize(numberofbadeles);
		thrust::fill_n(t_threadmarker.begin(), numberofencsegs, 0);
		thrust::fill_n(t_threadmarker.begin() + numberofencsegs, numberofencsubfaces, 1);
		thrust::fill_n(t_threadmarker.begin() + numberofencsegs + numberofencsubfaces, numberofbadtets, 2);

		//int numberofblocks = (ceil)((float)numberofbadeles / BLOCK_SIZE);
		//kernelCheckBadElementList << <numberofblocks, BLOCK_SIZE >> > (
		//	thrust::raw_pointer_cast(&t_badeleidlist[0]),
		//	thrust::raw_pointer_cast(&t_threadmarker[0]),
		//	thrust::raw_pointer_cast(&t_segencmarker[0]),
		//	thrust::raw_pointer_cast(&t_subfaceencmarker[0]),
		//	thrust::raw_pointer_cast(&t_tetstatus[0]),
		//	numberofencsegs,
		//	numberofencsubfaces,
		//	numberofbadtets,
		//	numberofbadeles
		//	);


		//tv[0] = clock();
		//npt[0] = numofpoints;
		// Insert points concurrently
		code =
			insertPoint(
				t_pointlist,
				t_point2trilist,
				t_point2tetlist,
				t_pointtypelist,
				t_pointradius,
				t_seglist,
				t_seg2trilist,
				t_seg2tetlist,
				t_seg2parentidxlist,
				t_segparentendpointidxlist,
				t_segstatus,
				t_trifacelist,
				t_tri2tetlist,
				t_tri2trilist,
				t_tri2seglist,
				t_tri2parentidxlist,
				t_triid2parentoffsetlist,
				t_triparentendpointidxlist,
				t_tristatus,
				t_tetlist,
				t_neighborlist,
				t_tet2trilist,
				t_tet2seglist,
				t_tetstatus,
				t_segencmarker,
				t_subfaceencmarker,
				t_badeleidlist,
				t_threadmarker,
				numberofbadeles,
				numberofencsegs,
				numberofencsubfaces,
				numberofbadtets,
				numofpoints,
				numofsubseg,
				numofsubface,
				numoftet,
				behavior,
				iteration,
				-1,
				-1,
				debug_msg,
				debug_error,
				debug_timing
			);

		if (!code)
			break;

		hipDeviceSynchronize();

		iteration++;

		//tv[1] = clock();
		//npt[1] = numofpoints;
		//printf("%f, %d\n", (REAL)(tv[1] - tv[0]), npt[1] - npt[0]);
	}
	if (!code && debug_msg)
		printf("      End with %d bad tets\n", numberofbadtets);
}
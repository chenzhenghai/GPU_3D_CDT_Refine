#include <stdio.h>
#include <sstream>
#include <hip/hip_runtime.h>

#include "CudaAnimation.h"
#include "Mesh.h"

void outputStartingFrame(
	internalmesh* drawmesh,
	RealD& t_pointlist,
	IntD& t_tetlist,
	TetStatusD& t_tetstatus,
	IntD& t_threadlist,
	IntD& t_insertidxlist,
	RealD& t_insertptlist,
	TetHandleD& t_locatedtet,
	int iter_seg,
	int iter_subface,
	int iter_tet
)
{
	// Prepare lists
	int numberofpoints = t_pointlist.size() / 3;
	drawmesh->pointlist = new REAL[3 * numberofpoints];
	hipMemcpy(drawmesh->pointlist, thrust::raw_pointer_cast(&t_pointlist[0]), 3 * numberofpoints * sizeof(double), hipMemcpyDeviceToHost);

	int numberoftet = t_tetstatus.size();
	drawmesh->tetlist = new int[4 * numberoftet];
	hipMemcpy(drawmesh->tetlist, thrust::raw_pointer_cast(&t_tetlist[0]), 4 * numberoftet * sizeof(int), hipMemcpyDeviceToHost);
	drawmesh->tetstatus = new tetstatus[numberoftet];
	hipMemcpy(drawmesh->tetstatus, thrust::raw_pointer_cast(&t_tetstatus[0]), numberoftet * sizeof(tetstatus), hipMemcpyDeviceToHost);

	int numberofthreads = t_threadlist.size();
	drawmesh->threadlist = new int[numberofthreads];
	hipMemcpy(drawmesh->threadlist, thrust::raw_pointer_cast(&t_threadlist[0]), numberofthreads * sizeof(int), hipMemcpyDeviceToHost);

	int numofinsertpt = t_insertidxlist.size();
	drawmesh->insertidxlist = new int[numofinsertpt];
	hipMemcpy(drawmesh->insertidxlist, thrust::raw_pointer_cast(&t_insertidxlist[0]), numofinsertpt * sizeof(int), hipMemcpyDeviceToHost);
	drawmesh->insertptlist = new REAL[3 * numofinsertpt];
	hipMemcpy(drawmesh->insertptlist, thrust::raw_pointer_cast(&t_insertptlist[0]), 3 * numofinsertpt * sizeof(double), hipMemcpyDeviceToHost);
	drawmesh->locatedtet = new tethandle[numofinsertpt];
	hipMemcpy(drawmesh->locatedtet, thrust::raw_pointer_cast(&t_locatedtet[0]), numofinsertpt * sizeof(tethandle), hipMemcpyDeviceToHost);

	// Write to mesh file
	// First frame: whole triangulation with bad tetrahedra colored
	{
		// filename
		FILE * fp;
		char *file;
		std::ostringstream strs;
		std::string fn;
		strs << "animation/F000_" << iter_seg << "_" << iter_subface << "_" << iter_tet << ".mesh";
		fn = strs.str();
		file = new char[fn.length() + 1];
		strcpy(file, fn.c_str());

		// writing
		fp = fopen(file, "w");
		fprintf(fp, "MeshVersionFormatted 1\n");
		fprintf(fp, "Dimension 3\n");

		fprintf(fp, "Vertices\n");
		fprintf(fp, "%d\n", numberofpoints);
		for (int i = 0; i < numberofpoints; i++)
			fprintf(fp, "%lf %lf %lf 1\n", drawmesh->pointlist[3 * i + 0], drawmesh->pointlist[3 * i + 1], drawmesh->pointlist[3 * i + 2]);

		// Color the faces of bad tetrahedra
		int numberoftriface = 0;
		for (int i = 0; i < numberoftet; i++)
		{
			if (!drawmesh->tetstatus[i].isEmpty())
				numberoftriface += 4;
		}
		numberoftriface += 4 * numberofthreads;
		fprintf(fp, "Triangles\n");
		fprintf(fp, "%d\n", numberoftriface);
		for (int i = 0; i < numberoftet; i++)
		{
			if (!drawmesh->tetstatus[i].isEmpty())
			{
				for (int j = 0; j < 4; j++)
					fprintf(fp, "%d %d %d 1\n", drawmesh->tetlist[4 * i + (j + 1) % 4] + 1,
						drawmesh->tetlist[4 * i + (j + 2) % 4] + 1, drawmesh->tetlist[4 * i + (j + 3) % 4] + 1);
			}
		}
		int threadIdx, tetIdx, colorIdx;
		for (int i = 0; i < numberofthreads; i++)
		{
			threadIdx = drawmesh->threadlist[i];
			colorIdx = (threadIdx % 32) + 2;
			tetIdx = drawmesh->insertidxlist[threadIdx];
			for (int j = 0; j < 4; j++)
				fprintf(fp, "%d %d %d %d\n", drawmesh->tetlist[4 * tetIdx + (j + 1) % 4] + 1,
					drawmesh->tetlist[4 * tetIdx + (j + 2) % 4] + 1, drawmesh->tetlist[4 * tetIdx + (j + 3) % 4] + 1, colorIdx);
		}

		fprintf(fp, "Tetrahedra\n");
		fprintf(fp, "%d\n", numberoftet);
		for (int i = 0; i < numberoftet; i++)
		{
			if (!drawmesh->tetstatus[i].isEmpty())
				fprintf(fp, "%d %d %d %d 1\n", drawmesh->tetlist[4 * i + 0] + 1, drawmesh->tetlist[4 * i + 1] + 1,
					drawmesh->tetlist[4 * i + 2] + 1, drawmesh->tetlist[4 * i + 3] + 1);
		}
		fprintf(fp, "End\n");
		fclose(fp);
		delete[] file;
	}
	// First frame info
	{
		// filename
		FILE * fp;
		char *file;
		std::ostringstream strs;
		std::string fn;
		strs << "animation/F000_" << iter_seg << "_" << iter_subface << "_" << iter_tet << ".txt";
		fn = strs.str();
		file = new char[fn.length() + 1];
		strcpy(file, fn.c_str());

		// writing
		fp = fopen(file, "w");
		fprintf(fp, "Number of points = %d\n", numberofpoints);
		fprintf(fp, "Number of tets = %d\n", numberoftet);
		fprintf(fp, "Number of bad tets = %d\n", numberofthreads);
		fclose(fp);
	}

	// Second frame: bad tetrahedra colored
	{
		// filename
		FILE * fp;
		char *file;
		std::ostringstream strs;
		std::string fn;
		strs << "animation/F001_" << iter_seg << "_" << iter_subface << "_" << iter_tet << ".mesh";
		fn = strs.str();
		file = new char[fn.length() + 1];
		strcpy(file, fn.c_str());

		// writing
		fp = fopen(file, "w");
		fprintf(fp, "MeshVersionFormatted 1\n");
		fprintf(fp, "Dimension 3\n");

		fprintf(fp, "Vertices\n");
		fprintf(fp, "%d\n", numberofpoints + numberofthreads);
		for (int i = 0; i < numberofpoints; i++)
			fprintf(fp, "%lf %lf %lf 1\n", drawmesh->pointlist[3 * i + 0], drawmesh->pointlist[3 * i + 1], drawmesh->pointlist[3 * i + 2]);
		int threadIdx, colorIdx;
		for (int i = 0; i < numberofthreads; i++)
		{
			threadIdx = drawmesh->threadlist[i];
			colorIdx = (threadIdx % 32) + 2;
			fprintf(fp, "%lf %lf %lf %d\n", drawmesh->insertptlist[3 * threadIdx + 0], drawmesh->insertptlist[3 * threadIdx + 1],
				drawmesh->insertptlist[3 * threadIdx + 2], colorIdx);
		}

		// Color the faces of bad tetrahedra
		int numberoftriface = 4 * numberofthreads;
		fprintf(fp, "Triangles\n");
		fprintf(fp, "%d\n", numberoftriface);
		int tetIdx;
		for (int i = 0; i < numberofthreads; i++)
		{
			threadIdx = drawmesh->threadlist[i];
			colorIdx = (threadIdx % 32) + 2;
			tetIdx = drawmesh->insertidxlist[threadIdx];
			for (int j = 0; j < 4; j++)
				fprintf(fp, "%d %d %d %d\n", drawmesh->tetlist[4 * tetIdx + (j + 1) % 4] + 1,
					drawmesh->tetlist[4 * tetIdx + (j + 2) % 4] + 1, drawmesh->tetlist[4 * tetIdx + (j + 3) % 4] + 1, colorIdx);
		}

		fprintf(fp, "Tetrahedra\n");
		fprintf(fp, "0\n");
		fprintf(fp, "End\n");
		fclose(fp);
		delete[] file;
	}

	// Third frame: located tetrahedra colored
	{
		// filename
		FILE * fp;
		char *file;
		std::ostringstream strs;
		std::string fn;
		strs << "animation/F002_" << iter_seg << "_" << iter_subface << "_" << iter_tet << ".mesh";
		fn = strs.str();
		file = new char[fn.length() + 1];
		strcpy(file, fn.c_str());

		// writing
		fp = fopen(file, "w");
		fprintf(fp, "MeshVersionFormatted 1\n");
		fprintf(fp, "Dimension 3\n");

		fprintf(fp, "Vertices\n");
		fprintf(fp, "%d\n", numberofpoints);
		for (int i = 0; i < numberofpoints; i++)
			fprintf(fp, "%lf %lf %lf 1\n", drawmesh->pointlist[3 * i + 0], drawmesh->pointlist[3 * i + 1], drawmesh->pointlist[3 * i + 2]);

		int numberoftriface = 4 * numberofthreads;
		fprintf(fp, "Triangles\n");
		fprintf(fp, "%d\n", numberoftriface);
		int threadIdx, colorIdx, tetIdx;
		for (int i = 0; i < numberofthreads; i++)
		{
			threadIdx = drawmesh->threadlist[i];
			colorIdx = (threadIdx % 32) + 2;
			tetIdx = drawmesh->locatedtet[threadIdx].id;
			for (int j = 0; j < 4; j++)
				fprintf(fp, "%d %d %d %d\n", drawmesh->tetlist[4 * tetIdx + (j + 1) % 4] + 1,
					drawmesh->tetlist[4 * tetIdx + (j + 2) % 4] + 1, drawmesh->tetlist[4 * tetIdx + (j + 3) % 4] + 1, colorIdx);
		}

		fprintf(fp, "Tetrahedra\n");
		fprintf(fp, "0\n");
		fprintf(fp, "End\n");
		fclose(fp);
		delete[] file;
	}
	// Third frame info
	{
		// filename
		FILE * fp;
		char *file;
		std::ostringstream strs;
		std::string fn;
		strs << "animation/F002_" << iter_seg << "_" << iter_subface << "_" << iter_tet << ".txt";
		fn = strs.str();
		file = new char[fn.length() + 1];
		strcpy(file, fn.c_str());

		bool* flag = new bool[numofinsertpt];
		for (int i = 0; i < numofinsertpt; i++)
			flag[i] = false;
		for (int i = 0; i < numberofthreads; i++)
		{
			int threadIdx = drawmesh->threadlist[i];;
			flag[threadIdx] = true;
		}
		for (int i = 0; i < numofinsertpt; i++)
		{
			if (flag[i])
			{
				int id1 = drawmesh->locatedtet[i].id;
				for (int j = i + 1; j < numofinsertpt; j++)
				{
					if (flag[j])
					{
						int id2 = drawmesh->locatedtet[j].id;
						if (id1 == id2)
							flag[j] = false;
					}
				}
			}
		}
		numberoftet = 0;
		for (int i = 0; i < numofinsertpt; i++)
		{
			if (flag[i])
				numberoftet++;
		}
		delete[] flag;

		// writing
		fp = fopen(file, "w");
		fprintf(fp, "Number of located tets = %d\n", numberoftet);
		fclose(fp);
	}

	// Update frame number
	drawmesh->framenum = 3;

	// Clear memory
	delete[] drawmesh->pointlist;
	delete[] drawmesh->tetlist;
	delete[] drawmesh->tetstatus;
	delete[] drawmesh->threadlist;
	delete[] drawmesh->insertidxlist;
}

void outputCavityFrame(
	internalmesh* drawmesh,
	RealD& t_pointlist,
	IntD& t_tetlist,
	UInt64D& t_tetmarker,
	IntD& t_threadmarker,
	TetHandleD& t_caveoldtetlist,
	IntD& t_caveoldtetnext,
	IntD& t_caveoldtethead,
	int iter_seg,
	int iter_subface,
	int iter_tet,
	int iter_expanding,
	int expandingsize
)
{
	// Prepare lists
	int numberofpoints = t_pointlist.size() / 3;
	drawmesh->pointlist = new REAL[3 * numberofpoints];
	hipMemcpy(drawmesh->pointlist, thrust::raw_pointer_cast(&t_pointlist[0]), 3 * numberofpoints * sizeof(double), hipMemcpyDeviceToHost);

	drawmesh->tetlist = new int[t_tetlist.size()];
	hipMemcpy(drawmesh->tetlist, thrust::raw_pointer_cast(&t_tetlist[0]), t_tetlist.size() * sizeof(int), hipMemcpyDeviceToHost);
	drawmesh->tetmarker = new unsigned long long[t_tetmarker.size()];
	hipMemcpy(drawmesh->tetmarker, thrust::raw_pointer_cast(&t_tetmarker[0]), t_tetmarker.size() * sizeof(unsigned long long), hipMemcpyDeviceToHost);

	int numofinsertpt = t_threadmarker.size();
	drawmesh->threadmarker = new int[numofinsertpt];
	hipMemcpy(drawmesh->threadmarker, thrust::raw_pointer_cast(&t_threadmarker[0]), numofinsertpt * sizeof(int), hipMemcpyDeviceToHost);
	drawmesh->caveoldtethead = new int[numofinsertpt];
	hipMemcpy(drawmesh->caveoldtethead, thrust::raw_pointer_cast(&t_caveoldtethead[0]), numofinsertpt * sizeof(int), hipMemcpyDeviceToHost);

	drawmesh->caveoldtetlist = new tethandle[t_caveoldtetlist.size()];
	hipMemcpy(drawmesh->caveoldtetlist, thrust::raw_pointer_cast(&t_caveoldtetlist[0]), t_caveoldtetlist.size() * sizeof(tethandle), hipMemcpyDeviceToHost);
	drawmesh->caveoldtetnext = new int[t_caveoldtetnext.size()];
	hipMemcpy(drawmesh->caveoldtetnext, thrust::raw_pointer_cast(&t_caveoldtetnext[0]), t_caveoldtetnext.size() * sizeof(int), hipMemcpyDeviceToHost);

	// frame
	int numoftet = 0, numofhulltet = 0, numofsp = 0;
	{
		// filename
		FILE * fp;
		char *file;
		std::ostringstream strs;
		std::string fn;
		if(drawmesh->framenum < 10)
			strs << "animation/F00" << drawmesh->framenum << "_";
		else if(drawmesh->framenum < 100)
			strs << "animation/F0" << drawmesh->framenum << "_";
		else
			strs << "animation/F" << drawmesh->framenum << "_";

		strs << iter_seg << "_" << iter_subface << "_" << iter_tet << "_" << iter_expanding << ".mesh";
		fn = strs.str();
		file = new char[fn.length() + 1];
		strcpy(file, fn.c_str());

		// writing
		fp = fopen(file, "w");
		fprintf(fp, "MeshVersionFormatted 1\n");
		fprintf(fp, "Dimension 3\n");

		fprintf(fp, "Vertices\n");
		fprintf(fp, "%d\n", numberofpoints);
		for (int i = 0; i < numberofpoints; i++)
			fprintf(fp, "%lf %lf %lf 1\n", drawmesh->pointlist[3 * i + 0], drawmesh->pointlist[3 * i + 1], drawmesh->pointlist[3 * i + 2]);
		int threadIdx, tetIdx;
		for (int i = 0; i < numofinsertpt; i++)
		{
			if (drawmesh->threadmarker[i] != -1)
			{
				threadIdx = i;
				numofsp++;
				int j = drawmesh->caveoldtethead[i];
				while (j != -1)
				{
					tetIdx = drawmesh->caveoldtetlist[j].id;
					if( (drawmesh->tetmarker[tetIdx] & 0xFFFFFFFF) == threadIdx/* && drawmesh->tetlist[4*tetIdx + 3] != -1*/)
						numoftet++;
					if ((drawmesh->tetmarker[tetIdx] & 0xFFFFFFFF) == threadIdx && drawmesh->tetlist[4 * tetIdx + 3] == -1)
						numofhulltet++;
					j = drawmesh->caveoldtetnext[j];
				}
			}
		}
		int numberoftriface = 4 * numoftet;
		fprintf(fp, "Triangles\n");
		fprintf(fp, "%d\n", numberoftriface);
		int colorIdx;
		for (int i = 0; i < numofinsertpt; i++)
		{
			if (drawmesh->threadmarker[i] != -1)
			{
				threadIdx = i;
				colorIdx = (threadIdx % 32) + 2;

				int k = drawmesh->caveoldtethead[i];
				while (k != -1)
				{
					tetIdx = drawmesh->caveoldtetlist[k].id;
					if ((drawmesh->tetmarker[tetIdx] & 0xFFFFFFFF) == threadIdx /*&& drawmesh->tetlist[4 * tetIdx + 3] != -1*/)
					{
						for (int j = 0; j < 4; j++)
							fprintf(fp, "%d %d %d %d\n", drawmesh->tetlist[4 * tetIdx + (j + 1) % 4] + 1,
								drawmesh->tetlist[4 * tetIdx + (j + 2) % 4] + 1, drawmesh->tetlist[4 * tetIdx + (j + 3) % 4] + 1, colorIdx);
					}

					k = drawmesh->caveoldtetnext[k];
				}
			}
		}

		fprintf(fp, "Tetrahedra\n");
		fprintf(fp, "0\n");
		fprintf(fp, "End\n");
		fclose(fp);
		delete[] file;
	}

	// frame info
	{
		// filename
		FILE * fp;
		char *file;
		std::ostringstream strs;
		std::string fn;
		if (drawmesh->framenum < 10)
			strs << "animation/F00" << drawmesh->framenum << "_";
		else if (drawmesh->framenum < 100)
			strs << "animation/F0" << drawmesh->framenum << "_";
		else
			strs << "animation/F" << drawmesh->framenum << "_";

		strs << iter_seg << "_" << iter_subface << "_" << iter_tet << "_" << iter_expanding << ".txt";
		fn = strs.str();
		file = new char[fn.length() + 1];
		strcpy(file, fn.c_str());

		// writing
		fp = fopen(file, "w");
		fprintf(fp, "Number of splitting points = %d\n", numofsp);
		fprintf(fp, "Number of cavity tets = %d\n", numoftet);
		fprintf(fp, "Number of hull tets = %d\n", numofhulltet);
		fprintf(fp, "Number of expanded tets = %d\n", expandingsize);
		fclose(fp);
		delete[] file;
	}

	drawmesh->framenum++;

	// Clear memory
	delete[] drawmesh->pointlist;
	delete[] drawmesh->tetlist;
	delete[] drawmesh->tetmarker;
	delete[] drawmesh->threadmarker;
	delete[] drawmesh->caveoldtetlist;
	delete[] drawmesh->caveoldtetnext;
	delete[] drawmesh->caveoldtethead;
}

void outputCavityFrame(
	internalmesh* drawmesh,
	RealD& t_pointlist,
	IntD& t_trifacelist,
	TriStatusD& t_tristatus,
	TetHandleD& t_tri2tetlist,
	IntD& t_tetlist,
	TetHandleD& t_neighborlist,
	UInt64D& t_tetmarker,
	IntD& t_threadmarker,
	TetHandleD& t_caveoldtetlist,
	IntD& t_caveoldtetnext,
	IntD& t_caveoldtethead,
	int iter_seg,
	int iter_subface,
	int iter_tet,
	int iter_expanding,
	int expandingsize
)
{
	// Prepare lists
	int numberofpoints = t_pointlist.size() / 3;
	drawmesh->pointlist = new REAL[3 * numberofpoints];
	hipMemcpy(drawmesh->pointlist, thrust::raw_pointer_cast(&t_pointlist[0]), 3 * numberofpoints * sizeof(double), hipMemcpyDeviceToHost);

	int numofsubface = t_tristatus.size();
	drawmesh->numofsubface = numofsubface;
	drawmesh->trifacelist = new int[3 * numofsubface];
	hipMemcpy(drawmesh->trifacelist, thrust::raw_pointer_cast(&t_trifacelist[0]), 3 * numofsubface * sizeof(int), hipMemcpyDeviceToHost);
	drawmesh->tristatus = new tristatus[numofsubface];
	hipMemcpy(drawmesh->tristatus, thrust::raw_pointer_cast(&t_tristatus[0]), numofsubface * sizeof(tristatus), hipMemcpyDeviceToHost);
	drawmesh->tri2tetlist = new tethandle[2 * numofsubface];
	hipMemcpy(drawmesh->tri2tetlist, thrust::raw_pointer_cast(&t_tri2tetlist[0]), 2 * numofsubface * sizeof(tethandle), hipMemcpyDeviceToHost);

	drawmesh->tetlist = new int[t_tetlist.size()];
	hipMemcpy(drawmesh->tetlist, thrust::raw_pointer_cast(&t_tetlist[0]), t_tetlist.size() * sizeof(int), hipMemcpyDeviceToHost);
	drawmesh->tetmarker = new unsigned long long[t_tetmarker.size()];
	hipMemcpy(drawmesh->tetmarker, thrust::raw_pointer_cast(&t_tetmarker[0]), t_tetmarker.size() * sizeof(unsigned long long), hipMemcpyDeviceToHost);
	drawmesh->neighborlist = new tethandle[t_neighborlist.size()];
	hipMemcpy(drawmesh->neighborlist, thrust::raw_pointer_cast(&t_neighborlist[0]), t_neighborlist.size() * sizeof(tethandle), hipMemcpyDeviceToHost);

	int numofinsertpt = t_threadmarker.size();
	drawmesh->threadmarker = new int[numofinsertpt];
	hipMemcpy(drawmesh->threadmarker, thrust::raw_pointer_cast(&t_threadmarker[0]), numofinsertpt * sizeof(int), hipMemcpyDeviceToHost);
	drawmesh->caveoldtethead = new int[numofinsertpt];
	hipMemcpy(drawmesh->caveoldtethead, thrust::raw_pointer_cast(&t_caveoldtethead[0]), numofinsertpt * sizeof(int), hipMemcpyDeviceToHost);

	drawmesh->caveoldtetlist = new tethandle[t_caveoldtetlist.size()];
	hipMemcpy(drawmesh->caveoldtetlist, thrust::raw_pointer_cast(&t_caveoldtetlist[0]), t_caveoldtetlist.size() * sizeof(tethandle), hipMemcpyDeviceToHost);
	drawmesh->caveoldtetnext = new int[t_caveoldtetnext.size()];
	hipMemcpy(drawmesh->caveoldtetnext, thrust::raw_pointer_cast(&t_caveoldtetnext[0]), t_caveoldtetnext.size() * sizeof(int), hipMemcpyDeviceToHost);

	// frame
	int numoftet = 0, numofhulltet = 0, numofsp = 0, numofcs = 0, numofttcs = 0;;
	{
		// filename
		FILE * fp;
		char *file;
		std::ostringstream strs;
		std::string fn;
		if (drawmesh->framenum < 10)
			strs << "animation/F00" << drawmesh->framenum << "_";
		else if (drawmesh->framenum < 100)
			strs << "animation/F0" << drawmesh->framenum << "_";
		else
			strs << "animation/F" << drawmesh->framenum << "_";

		strs << iter_seg << "_" << iter_subface << "_" << iter_tet << "_" << iter_expanding << ".mesh";
		fn = strs.str();
		file = new char[fn.length() + 1];
		strcpy(file, fn.c_str());

		// writing
		fp = fopen(file, "w");
		fprintf(fp, "MeshVersionFormatted 1\n");
		fprintf(fp, "Dimension 3\n");

		fprintf(fp, "Vertices\n");
		fprintf(fp, "%d\n", numberofpoints);
		for (int i = 0; i < numberofpoints; i++)
			fprintf(fp, "%lf %lf %lf 1\n", drawmesh->pointlist[3 * i + 0], drawmesh->pointlist[3 * i + 1], drawmesh->pointlist[3 * i + 2]);
		int threadIdx, tetIdx;
		for (int i = 0; i < numofinsertpt; i++)
		{
			if (drawmesh->threadmarker[i] != -1)
			{
				threadIdx = i;
				numofsp++;
				int j = drawmesh->caveoldtethead[i];
				while (j != -1)
				{
					tetIdx = drawmesh->caveoldtetlist[j].id;
					if ((drawmesh->tetmarker[tetIdx] & 0xFFFFFFFF) == threadIdx && drawmesh->tetlist[4 * tetIdx + 3] != -1)
						numoftet++;
					if ((drawmesh->tetmarker[tetIdx] & 0xFFFFFFFF) == threadIdx && drawmesh->tetlist[4 * tetIdx + 3] == -1)
						numofhulltet++;
					j = drawmesh->caveoldtetnext[j];
				}
			}
		}

		int numberoftriface = 4 * numoftet;
		for (int i = 0; i < numofsubface; i++)
		{
			if (!drawmesh->tristatus[i].isEmpty())
			{
				numofttcs++;
				trihandle checksh(i, 0);
				tethandle checktet;
				stpivot(checksh, checktet, drawmesh->tri2tetlist);
				if (ishulltet(checktet, drawmesh->tetlist))
					continue;
				fsymself(checktet, drawmesh->neighborlist);
				if (ishulltet(checktet, drawmesh->tetlist))
					continue;
				numberoftriface++;
			}
		}

		fprintf(fp, "Triangles\n");
		fprintf(fp, "%d\n", numberoftriface);
		int colorIdx;
		for (int i = 0; i < numofinsertpt; i++)
		{
			if (drawmesh->threadmarker[i] != -1)
			{
				threadIdx = i;
				colorIdx = (threadIdx % 32) + 2;

				int k = drawmesh->caveoldtethead[i];
				while (k != -1)
				{
					tetIdx = drawmesh->caveoldtetlist[k].id;
					if ((drawmesh->tetmarker[tetIdx] & 0xFFFFFFFF) == threadIdx && drawmesh->tetlist[4 * tetIdx + 3] != -1)
					{
						for (int j = 0; j < 4; j++)
							fprintf(fp, "%d %d %d %d\n", drawmesh->tetlist[4 * tetIdx + (j + 1) % 4] + 1,
								drawmesh->tetlist[4 * tetIdx + (j + 2) % 4] + 1, drawmesh->tetlist[4 * tetIdx + (j + 3) % 4] + 1, colorIdx);
					}

					k = drawmesh->caveoldtetnext[k];
				}
			}
		}
		for (int i = 0; i < numofsubface; i++)
		{
			if (!drawmesh->tristatus[i].isEmpty())
			{
				trihandle checksh(i, 0);
				tethandle checktet;
				stpivot(checksh, checktet, drawmesh->tri2tetlist);
				if (ishulltet(checktet, drawmesh->tetlist))
					continue;
				fsymself(checktet, drawmesh->neighborlist);
				if (ishulltet(checktet, drawmesh->tetlist))
					continue;
				fprintf(fp, "%d %d %d 1\n", drawmesh->trifacelist[3 * i] + 1, drawmesh->trifacelist[3 * i + 1] + 1,
					drawmesh->trifacelist[3 * i + 2] + 1);
			}
		}

		fprintf(fp, "Tetrahedra\n");
		fprintf(fp, "0\n");
		fprintf(fp, "End\n");
		fclose(fp);
		delete[] file;
	}

	// frame info
	{
		// filename
		FILE * fp;
		char *file;
		std::ostringstream strs;
		std::string fn;
		if (drawmesh->framenum < 10)
			strs << "animation/F00" << drawmesh->framenum << "_";
		else if (drawmesh->framenum < 100)
			strs << "animation/F0" << drawmesh->framenum << "_";
		else
			strs << "animation/F" << drawmesh->framenum << "_";

		strs << iter_seg << "_" << iter_subface << "_" << iter_tet << "_" << iter_expanding << ".txt";
		fn = strs.str();
		file = new char[fn.length() + 1];
		strcpy(file, fn.c_str());

		// writing
		fp = fopen(file, "w");
		fprintf(fp, "Number of splitting points = %d\n", numofsp);
		fprintf(fp, "Number of cavity tets = %d\n", numoftet);
		fprintf(fp, "Number of hull tets = %d\n", numofhulltet);
		fprintf(fp, "Number of expanded tets = %d\n", expandingsize);
		fprintf(fp, "Number of subfaces = %d\n", numofttcs);
		fclose(fp);
		delete[] file;
	}

	drawmesh->framenum++;

	// Clear memory
	delete[] drawmesh->pointlist;
	delete[] drawmesh->tetlist;
	delete[] drawmesh->tetmarker;
	delete[] drawmesh->threadmarker;
	delete[] drawmesh->caveoldtetlist;
	delete[] drawmesh->caveoldtetnext;
	delete[] drawmesh->caveoldtethead;
}

void outputTmpMesh(
	internalmesh* drawmesh,
	RealD& t_pointlist,
	PointTypeD& t_pointtypelist,
	IntD& t_seglist,
	TriStatusD& t_segstatus,
	IntD& t_trifacelist,
	TriStatusD& t_tristatus,
	IntD& t_tetlist,
	TetStatusD& t_tetstatus,
	IntD& t_insertidxlist,
	RealD& t_insertptlist,
	IntD& t_threadlist,
	IntD& t_threadmarker,
	TetHandleD& t_cavebdrylist,
	IntD& t_cavebdrynext,
	IntD& t_cavebdryhead,
	int insertiontype
)
{
	int numofpoints = t_pointtypelist.size();
	drawmesh->numofpoints = numofpoints;
	drawmesh->pointlist = new REAL[3 * numofpoints];
	hipMemcpy(drawmesh->pointlist, thrust::raw_pointer_cast(&t_pointlist[0]), 3 * numofpoints * sizeof(double), hipMemcpyDeviceToHost);
	drawmesh->pointtype = new verttype[numofpoints];
	hipMemcpy(drawmesh->pointtype, thrust::raw_pointer_cast(&t_pointtypelist[0]), numofpoints * sizeof(verttype), hipMemcpyDeviceToHost);

	int numofsubseg = t_segstatus.size();
	drawmesh->numofsubseg = numofsubseg;
	drawmesh->seglist = new int[3 * numofsubseg];
	hipMemcpy(drawmesh->seglist, thrust::raw_pointer_cast(&t_seglist[0]), 3 * numofsubseg * sizeof(int), hipMemcpyDeviceToHost);
	drawmesh->segstatus = new tristatus[numofsubseg];
	hipMemcpy(drawmesh->segstatus, thrust::raw_pointer_cast(&t_segstatus[0]), numofsubseg * sizeof(tristatus), hipMemcpyDeviceToHost);

	int numofsubface = t_tristatus.size();
	drawmesh->numofsubface = numofsubface;
	drawmesh->trifacelist = new int[3 * numofsubface];
	hipMemcpy(drawmesh->trifacelist, thrust::raw_pointer_cast(&t_trifacelist[0]), 3 * numofsubface * sizeof(int), hipMemcpyDeviceToHost);
	drawmesh->tristatus = new tristatus[numofsubface];
	hipMemcpy(drawmesh->tristatus, thrust::raw_pointer_cast(&t_tristatus[0]), numofsubface * sizeof(tristatus), hipMemcpyDeviceToHost);

	int numoftet = t_tetstatus.size();
	drawmesh->numoftet = numoftet;
	drawmesh->tetlist = new int[4 * numoftet];
	hipMemcpy(drawmesh->tetlist, thrust::raw_pointer_cast(&t_tetlist[0]), 4 * numoftet * sizeof(int), hipMemcpyDeviceToHost);
	drawmesh->tetstatus = new tetstatus[numoftet];
	hipMemcpy(drawmesh->tetstatus, thrust::raw_pointer_cast(&t_tetstatus[0]), numoftet * sizeof(tetstatus), hipMemcpyDeviceToHost);

	// cavity
	int numberofthreads = t_threadlist.size();
	drawmesh->numofthread = numberofthreads;
	drawmesh->threadlist = new int[numberofthreads];
	hipMemcpy(drawmesh->threadlist, thrust::raw_pointer_cast(&t_threadlist[0]), numberofthreads * sizeof(int), hipMemcpyDeviceToHost);
	int numofinsertpt = t_insertidxlist.size();
	drawmesh->numofinsertpt = numofinsertpt;
	drawmesh->insertiontype = insertiontype;
	drawmesh->insertidxlist = new int[numofinsertpt];
	hipMemcpy(drawmesh->insertidxlist, thrust::raw_pointer_cast(&t_insertidxlist[0]), numofinsertpt * sizeof(int), hipMemcpyDeviceToHost);
	drawmesh->insertptlist = new REAL[3 * numofinsertpt];
	hipMemcpy(drawmesh->insertptlist, thrust::raw_pointer_cast(&t_insertptlist[0]), 3 * numofinsertpt * sizeof(double), hipMemcpyDeviceToHost);
	drawmesh->threadmarker = new int[numofinsertpt];
	hipMemcpy(drawmesh->threadmarker, thrust::raw_pointer_cast(&t_threadmarker[0]), numofinsertpt * sizeof(int), hipMemcpyDeviceToHost);
	drawmesh->cavebdrylist = new tethandle[t_cavebdrylist.size()];
	hipMemcpy(drawmesh->cavebdrylist, thrust::raw_pointer_cast(&t_cavebdrylist[0]), t_cavebdrylist.size() * sizeof(tethandle), hipMemcpyDeviceToHost);
	drawmesh->cavebdrynext = new int[t_cavebdrynext.size()];
	hipMemcpy(drawmesh->cavebdrynext, thrust::raw_pointer_cast(&t_cavebdrynext[0]), t_cavebdrynext.size() * sizeof(int), hipMemcpyDeviceToHost);
	drawmesh->cavebdryhead = new int[numofinsertpt];
	hipMemcpy(drawmesh->cavebdryhead, thrust::raw_pointer_cast(&t_cavebdryhead[0]), numofinsertpt * sizeof(int), hipMemcpyDeviceToHost);
}
